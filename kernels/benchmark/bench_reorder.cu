#include "reorder.cuh"

#include "cutlass/numeric_conversion.h"

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::nv_float4_t<cutlass::float_e2m1_t>;    // Element type for A matrix operand

// B matrix configuration
using         ElementB    = cutlass::nv_float4_t<cutlass::float_e2m1_t>;    // Element type for B matrix operand

using         ElementD    = cutlass::bfloat16_t;                            // Element type for D matrix operand
using         ElementC    = cutlass::bfloat16_t;                            // Element type for C matrix operand

int main() {
    
    const int M = 128;
    const int N = 4096;
    const int K = 4096;
    const int block_size = 16; 
    
    ElementA::DataType *A;
    ElementB::DataType *B;
    ElementC *X;
    ElementD *W;
    A = new ElementA::DataType[M * K];
    B = new ElementB::DataType[N * K];
    X = new ElementC[M * K];
    W = new ElementD[N * K];
    
    // 创建 scale 数组（每 block_size 个元素对应一个缩放因子）
    int szA = ((M * K + block_size - 1) / block_size);
    ElementA::ScaleFactorType *scaleA = new ElementA::ScaleFactorType[((M * K + block_size - 1) / block_size)];
    int szB = ((N * K + block_size - 1) / block_size);
    ElementB::ScaleFactorType *scaleB = new ElementB::ScaleFactorType[((N * K + block_size - 1) / block_size)];
    
    std::srand(static_cast<unsigned int>(std::time(0)));
    cutlass::NumericConverter<ElementC, float, cutlass::FloatRoundStyle::round_to_nearest> converterX;
    cutlass::NumericConverter<ElementD, float, cutlass::FloatRoundStyle::round_to_nearest> converterW;
    
    for (int i = 0; i < M * K; ++i) {
        // 模拟浮点值
        float f = static_cast<float>(std::rand()) / RAND_MAX * 2000000000.0f - 1000000000.0f;
        
        X[i] = converterX(f);
    }
    for (int i = 0; i < N * K; ++i) {
        // 模拟浮点值
        float f = static_cast<float>(std::rand()) / RAND_MAX * 2000000000.0f - 1000000000.0f;
        
        W[i] = converterW(f);
    }
    int16_t *reorder_index = new int16_t[K];
    for(int i = 0; i < K; i++) {
        reorder_index[i] = i;
    }
    std::random_shuffle(reorder_index, reorder_index + K);
    ElementA::DataType *A_d;
    ElementB::DataType *B_d;
    ElementC *X_d;
    ElementD *W_d;    
    int16_t *reorder_index_d;
    ElementA::ScaleFactorType *SFA_d;
    ElementB::ScaleFactorType *SFB_d;

    hipMalloc((void**)&A_d, M * K * sizeof(ElementA::DataType));
    hipMalloc((void**)&B_d, N * K * sizeof(ElementB::DataType));
    hipMalloc((void**)&X_d, M * K * sizeof(ElementC));
    hipMalloc((void**)&W_d, N * K * sizeof(ElementD));
    hipMalloc((void**)&reorder_index_d, K * sizeof(int16_t));
    hipMalloc((void**)&SFA_d, szA * sizeof(ElementA::ScaleFactorType));
    hipMalloc((void**)&SFB_d, szB * sizeof(ElementB::ScaleFactorType));
    hipMemcpy(X_d, X, M * K * sizeof(ElementC), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W, N * K * sizeof(ElementD), hipMemcpyHostToDevice);
    hipMemcpy(reorder_index_d, reorder_index, K * sizeof(int16_t), hipMemcpyHostToDevice);

    // Timing using CUDA events
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    for (int it = 0; it < 200; it ++) {
        run_reorder_bf16_mixed<32, K>(
            X_d, M, reorder_index_d, 
            reinterpret_cast<uint8_t*>(AN_d), reinterpret_cast<uint8_t*>(AS_d), reinterpret_cast<uint8_t*>(AO_d), 
            SFAN_d, SFAS_d, SFAO_d, KN, KS, KO
        );
        run_reorder_bf16_mixed<32, K>(
            W_d, N, reorder_index_d, 
            reinterpret_cast<uint8_t*>(BN_d), reinterpret_cast<uint8_t*>(BS_d), reinterpret_cast<uint8_t*>(BO_d), 
            SFBN_d, SFBS_d, SFBO_d, KN, KS, KO
        );
        // matmul_host(AN_d, BN_d, AS_d, BS_d, AO_d, BO_d, M, N, KN, KS, KO, C_d, D_d, SFAN_d, SFBN_d, SFAS_d, SFBS_d, SFAO_d, SFBO_d);
    }
    CHECK_CUDA(hipEventRecord(start));
    for (int it = 0; it < 400; it ++) {
        run_reorder_bf16_mixed<32, K>(
            X_d, M, reorder_index_d, 
            reinterpret_cast<uint8_t*>(AN_d), reinterpret_cast<uint8_t*>(AS_d), reinterpret_cast<uint8_t*>(AO_d), 
            SFAN_d, SFAS_d, SFAO_d, KN, KS, KO
        );
        // run_reorder_bf16_mixed<32, K>(
        //     W_d, N, reorder_index_d, 
        //     reinterpret_cast<uint8_t*>(BN_d), reinterpret_cast<uint8_t*>(BS_d), reinterpret_cast<uint8_t*>(BO_d), 
        //     SFBN_d, SFBS_d, SFBO_d, KN, KS, KO
        // );
        // matmul_host(AN_d, BN_d, AS_d, BS_d, AO_d, BO_d, M, N, KN, KS, KO, C_d, D_d, SFAN_d, SFBN_d, SFAS_d, SFBS_d, SFAO_d, SFBO_d);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    // CRITICAL: Synchronize and check for errors immediately after kernel launch
    hipError_t kernel_err = hipGetLastError(); // Check for asynchronous errors from the kernel
    if (kernel_err != hipSuccess) {
        std::cerr << "CUDA error after launching: "
                << hipGetErrorString(kernel_err) << std::endl;
        // Optionally, throw an exception to propagate the error to Python
        throw std::runtime_error(std::string("CUDA error in : ") + hipGetErrorString(kernel_err));
    }

    hipError_t sync_err = hipDeviceSynchronize(); // Wait for the kernel to complete and check for runtime errors
    if (sync_err != hipSuccess) {
        std::cerr << "CUDA error during/after kernel synchronization: "
                << hipGetErrorString(sync_err) << std::endl;
        throw std::runtime_error(std::string("CUDA sync error in kernel: ") + hipGetErrorString(sync_err));
    }
    std::cout << "kernel finished and synced successfully." << std::endl; std::cout.flush();

    std::printf("REORDER kernel completed in %.3f ms\n", milliseconds / 400);
    std::cout << "reorder finished." << std::endl;
    hipFree(AN_d);
    hipFree(BN_d);
    hipFree(AS_d);
    hipFree(BS_d);
    hipFree(AO_d);
    hipFree(BO_d);
    hipFree(X_d);
    hipFree(W_d);
    hipFree(SFAN_d);
    hipFree(SFBN_d);
    hipFree(SFAS_d);
    hipFree(SFBS_d);
    hipFree(SFAO_d);
    hipFree(SFBO_d);
    return 0; 
}